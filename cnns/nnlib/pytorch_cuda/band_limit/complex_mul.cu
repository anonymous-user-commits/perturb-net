#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>
#include <string>

namespace {

/**
The high pole in the tent for the FFT based convolution is the element-wise
complex multiplication between each input map and each filter. It is not the
cost of the FFT operation, which for an input map of size NxCxHxW is N * C *
H * log H * W * log W and for a filter bank of size FxCxHxW is
F * C * H * log H * W * log W * I. The cost of the inverse FFT is;
N*F*H*logH*W*logW*I. The total cost of FFT is: H*logH*W*logW*(N*C+F*C+N*F*I).
The cost of the convolution in the frequency
domain is: N * F * C * H * W * 3 (the minimum number of real multiplications
in complex multiplication of two numbers), which is greater than the cost of FFT since:
N * F * C * I >> logH*logW*(N*C+F*C+N*F*I). We can estimate it by omitting I=2.
and assuming X can be substituted for N, F, and C, H, W. X^3 >> 4X^2log^2X.
We can consider the first filter from the ResNet-18 for CIFAR10 with batch of
size 128. The cost of FFT operations is: 5*5*(128*3+64*3+128*64*2) = 424000.
The cost of the convolution in the frequency domain is:
128 * 64 * 3 * 32 * 32 * 3 = 75497472. The cost of convolution is about more
than two orders of magnitude greater.

Our method aims at saving memory and utilizing as many GPU threads as
possible. We fuse the elementwise complex multiplication with the summation
along a given channel in a thread execution path to limit the memory size (from
N * F * C * H * W * I to the actual size of the output: N * F * H * W * I),
and avoid any additional synchronization by focusing on computation
of a single output cell (of coordinates: [n,f,h,w] in an output map.

The total number of GPU thread blocks is N * F * max threads per block: number
of input maps (e.g. images) in the batch and number of filters in the filter
bank.
Each block of threads is used to compute a single output plane of size
(H x W x I), which corresponds to the f-th channel plane in the n-th output map.
It is obtained after a point-wise complex multiplication between an input map n
and a filter f, where both the input map and the filter are of size H x W x I.

Each thread in a block of threads drills through each channel in an input map
on the level of a given (H,W) coordinate.
For image n, we set its starting index n_idx from n*C*H*W*I and the last
coordinate for a given plane is at (n_idx+H*W*I) = n_idx + channel_size (a
single channel size).

We define the number of threads in the block as a raw_stride.
Once a thread finishes summing values for all the channels C in the (H,W)
coordinate, it moves (raw_stride*(*W*I) = stride) positions to the next (H,W)
coordinate to be computed or finishes its execution.

We use min(max_threads_in_block, H*W) threads per block.

Timing for running a single forward pass of ResNet-18:
global correlation time:  6.692555665969849
global fft time:  0.6110324859619141
global complex time:  6.235848426818848
global irfft time:  0.49118685722351074
global correlation time:  6.737519025802612

Running forward pass of ResNet-18 100 times:
rfft time:  8.698078155517578
complex multiply time:  81.90509557723999
irfft time:  7.067264080047607
complex correlation time:  89.14919781684875
total time with FFT based conv2D:  147.95940494537354
total time with pytorch conv2D: 44.175782918930054
pytorch speedup over cFFT for testing ResNet-18:  3.3493329414648696 X

With compression, energy preserved 90% in the signals:
rfft time:  6.97404146194458
preserve energy time total:  29.02157688140869
complex multiply time:  26.571154594421387
irfft time:  5.745055437088013
complex correlation time:  36.62075328826904
conv2D FFT time:  117.9371497631073
total time with pytorch conv2D: 42.78296375274658
pytorch speedup over cFFT for testing ResNet-18:  2.7566381432734652

Run forward pass for the whole ResNet-18 dataset:
total time with pytorch conv2D:  5.489983320236206
total time with FFT based conv2D:  285.2201008796692
pytorch speedup over cFFT for testing ResNet-18:  51.95281738440648

We also implemented the complex multiplication in C++ using torch library, but
there was almost no difference between the Python based version in PyTorch and
the C++ version using the Torch C++ library. However, the custom CUDA
implementation saves us a lot of memory and accelerates the computation by
about 10X.

conv2D_fft_benchmark.py .
cuda multiply time:  0.32549619674682617
pytorch multiply time:  3.791210651397705
cuda speedup is:  11.647480644287043
(the computation was executed 1000X for sizes N, C, H, W, I = 128, 3, 32, 32, 2,
and F = 16  # number of filter banks

Savings in memory from about 20% (for 3 channel input) to even 94% for 128 filter
banks and N, C, H, W, I = 64, 64, 32, 32, 2.
CUDA:
+-----------------------------------------------------------------------------+
| Processes:                                                       GPU Memory |
|  GPU       PID   Type   Process name                             Usage      |
|=============================================================================|
|    0     15764      C   /home/ady/anaconda3/bin/python3.6           8607MiB |
|    0     25840      C   /home/ady/anaconda3/bin/python3.6            565MiB |
+-----------------------------------------------------------------------------+

PyTorch:
+-----------------------------------------------------------------------------+
| Processes:                                                       GPU Memory |
|  GPU       PID   Type   Process name                             Usage      |
|=============================================================================|
|    0     15764      C   /home/ady/anaconda3/bin/python3.6           8607MiB |
|    0     25908      C   /home/ady/anaconda3/bin/python3.6            681MiB |
+-----------------------------------------------------------------------------+

CUDA:
+-----------------------------------------------------------------------------+
| Processes:                                                       GPU Memory |
|  GPU       PID   Type   Process name                             Usage      |
|=============================================================================|
|    0     15764      C   /home/ady/anaconda3/bin/python3.6           8607MiB |
|    0     26124      C   /home/ady/anaconda3/bin/python3.6            753MiB |

PyTorch:
+-----------------------------------------------------------------------------+
| Processes:                                                       GPU Memory |
|  GPU       PID   Type   Process name                             Usage      |
|=============================================================================|
|    0     15764      C   /home/ady/anaconda3/bin/python3.6           8607MiB |
|    0     26072      C   -                                          12977MiB |
+-----------------------------------------------------------------------------+
*/
template <typename scalar_t>
__device__ __forceinline__ void single_mul(
    scalar_t x_re,
    scalar_t x_im,
    scalar_t y_re,
    scalar_t y_im,
    scalar_t* out_re,
    scalar_t* out_im) {

    scalar_t uavc = x_re * (y_re + y_im);
    *out_re += uavc - (x_re + x_im) * y_im;
    *out_im += (x_im - x_re) * y_re + uavc;
}


template <typename scalar_t>
__device__ __forceinline__ void single_add(
    scalar_t x_re,
    scalar_t x_im,
    scalar_t y_re,
    scalar_t y_im,
    scalar_t* out_re,
    scalar_t* out_im) {

    *out_re += x_re + y_re;
    *out_im += x_im + y_im;
}


template <typename scalar_t>
__global__ void complex_mul_cuda_kernel(
    const scalar_t* __restrict__ x,
    const scalar_t* __restrict__ y,
    scalar_t* __restrict__ out,
    const int N, const int F, const int C, const int plane_size) {

    const int I = 2; // the last dimension for the complex number
    const int channel_size = plane_size * I;
    const int image_size = C * channel_size;  // size of the image from the batch

    const int n = blockIdx.x; // current index of an image/input map in the batch
    const int f = blockIdx.y; // current index of a filter from the filter bank

    // stride for the H*W map is equal to the number of threads declared in a block
    const int stride = blockDim.x * I; // we need H*W threads per plane, each deals with C channels and I numbers

    const int n_idx = n * image_size;  // start index in the batch for this input map
    const int f_idx = f * image_size;  // start index in the bank for this filter

    // find index for the output
    const int no_idx = n * (F * channel_size); // output index for the batch data point
    const int fo_idx = f * channel_size;       // output index for the filter/channel

    // Each H*W plane contains H*W*I elements in depth.
    // We linearize it and start from 0, move by #threads*I steps in outer loop.
    const int start_idx = threadIdx.x*I;

    // index in the input map
    int N_idx = n_idx + start_idx; // index across the first channel plane (in the input map n).
    const int last_N_idx = n_idx + plane_size * I;  // last index for the starting position to compute the sum through each channel for this pixel

    // index in the filter
    int F_idx = f_idx + start_idx; // index across the first channel plane (in the filter f).

    // index in the output, we compute cells on a flat plane (no channels)
    int O_idx = no_idx + fo_idx + start_idx;

    while (N_idx < last_N_idx - 1)  {
        int cN_idx = N_idx;  // current input n index across the channels
        int cF_idx = F_idx;  // current filter f index across the channels

        scalar_t out_re = 0;
        scalar_t out_im = 0;

        // If we have 512 channels - then it is rather inefficient loop
        for (int c = 0; c < C; ++c) {
//            printf("n:%d,N_idx:%d,f:%d,threadIdx.x:%d,cN_idx:%d,cF_idx:%d,last_N_idx:%d\n", n, N_idx, f, threadIdx.x, cN_idx, cF_idx, last_N_idx);
//            if (N_idx > N*C*H*W*I || F_idx > F*C*H*W*I)
//                printf("error out of bound\n");
//            if (x[cN_idx] > 1 || x[cN_idx + 1] > 1 || y[cF_idx] > 1 || y[cF_idx + 1] > 1) {
//                printf("n:%d,N_idx:%d,f:%d,threadIdx.x:%d,cN_idx:%d,cF_idx:%d,last_N_idx:%d,O_idx:%d,in_re:%f,in_im:%f,filter_re:%f,filter_im:%f. Error, the position cN_idx and cF_idx was already touched.\n", n, N_idx, f, threadIdx.x, cN_idx, cF_idx, last_N_idx, O_idx, x[cN_idx], x[cN_idx + 1], y[cF_idx], y[cF_idx + 1]);
//            }
            scalar_t x_re = x[cN_idx];
            scalar_t x_im = x[cN_idx + 1];
            scalar_t y_re = y[cF_idx];
            scalar_t y_im = y[cF_idx + 1];
            single_mul(x_re, x_im, y_re, y_im, &out_re, &out_im);
//            x[cN_idx] = cN_idx;
//            x[cN_idx + 1] = cN_idx + 1;
//            y[cF_idx] = cF_idx;
//            y[cF_idx + 1] = cF_idx + 1;
            cN_idx += channel_size;  // this is rather an inefficient strided memory access
            cF_idx += channel_size;  // this is rather an inefficient strided memory access
        }
//        if (out[O_idx] > 1 || out[O_idx + 1] > 1) {
//            printf("n:%d,N_idx:%d,f:%d,threadIdx.x:%d,cN_idx:%d,cF_idx:%d,last_N_idx:%d,O_idx:%d,re:%f,im:%f. Error, the position was already computed.\n", n, N_idx, f, threadIdx.x, cN_idx, cF_idx, last_N_idx, O_idx, out[O_idx], out[O_idx+1]);
//        } else {
//            printf("n:%d,N_idx:%d,f:%d,threadIdx.x:%d,cN_idx:%d,cF_idx:%d,last_N_idx:%d,O_idx:%d,re:%f,im:%f. Correct.\n", n, N_idx, f, threadIdx.x, cN_idx, cF_idx, last_N_idx, O_idx, out[O_idx], out[O_idx+1]);
//        }
        out[O_idx] = out_re;
        out[O_idx + 1] = out_im;

        N_idx += stride;
        F_idx += stride;
        O_idx += stride;
    }
}

} // namespace

void complex_mul_cuda(
    at::Tensor x,
    at::Tensor y,
    at::Tensor out,
    int threads = 1024) {

    const auto N = x.size(0);  // batch_size
    const auto F = y.size(0);  // filter_bank_size
    const auto C = x.size(1);  // number of channels

    int plane_size;
    const size_t dim_size = x.sizes().size();
    if (dim_size == 5) { // 2D data
        // dimensions: N, C, H, W, I
        const auto H = x.size(2);  // height of the matrix
        const auto W = x.size(3);  // width of the matrix
        plane_size = H * W;
    } else if (dim_size == 4) {
        // dimensions: N, C, L, I
        plane_size = x.size(2);
    } else {
         throw "Unexpected number of dimensions: " + std::to_string(dim_size);
    }

    const auto x_blocks = N;
    const auto y_blocks = F;
    const dim3 blocks(x_blocks, y_blocks);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(x.type(), "complex_mul_cuda",
    ([&] {
        complex_mul_cuda_kernel<scalar_t><<<blocks, threads>>>(
        x.data<scalar_t>(), y.data<scalar_t>(), out.data<scalar_t>(),
        N, F, C, plane_size);
    }));
}

//template <typename scalar_t>
//void complex_mul_stride_no_permute_cuda_pure(
//    at::Tensor x,
//    at::Tensor y,
//    at::Tensor out,
//    int threads = 1024) {
//
//    const auto N = x.size(0);  // batch_size
//    const auto F = y.size(0);  // filter_bank_size
//    const auto C = x.size(1);  // number of channels
//    const auto H = x.size(2);  // height of the matrix
//    const auto W = x.size(3);  // width of the matrix
//
//    const auto x_blocks = N;
//    const auto y_blocks = F;
//    const dim3 blocks(x_blocks, y_blocks);
//
//    // Run kernel on the GPU
//    complex_mul_cuda_kernel<scalar_t><<<blocks, 1024>>>(
//        x.data<scalar_t>(), y.data<scalar_t>(), out.data<scalar_t>(),
//        N, F, C, H, W);
//}

/**
Uncomment the pytorch related stuff.

Compile:
ady@skr-compute1:/tmp/pycharm_project_154/cnns/nnlib/pytorch_cuda/complex_mul_cuda$ nvcc complex_mul_kernel_stride_no_permute.cu -o complex_mul_profile.out
ady@skr-compute1:/tmp/pycharm_project_154/cnns/nnlib/pytorch_cuda/complex_mul_cuda$ nvprof ./complex_mul_profile.out

nvidia

/usr/local/cuda/bin/nvcc -I/local/ady/anaconda3/lib/python3.6/site-packages/torch/lib/include -I/local/ady/anaconda3/lib/python3.6/site-packages/torch/lib/include/torch/csrc/api/include -I/local/ady/anaconda3/lib/python3.6/site-packages/torch/lib/include/TH -I/local/ady/anaconda3/lib/python3.6/site-packages/torch/lib/include/THC -I/usr/local/cuda/include -I/local/ady/anaconda3/include/python3.6m -c complex_mul_kernel.cu -o complex_mul_kernel_stride_no_permute.out -std=c++11
nvcc -I/local/ady/anaconda3/lib/python3.6/site-packages/torch/lib/include -I/local/ady/anaconda3/lib/python3.6/site-packages/torch/lib/include/torch/csrc/api/include -I/local/ady/anaconda3/lib/python3.6/site-packages/torch/lib/include/TH -I/local/ady/anaconda3/lib/python3.6/site-packages/torch/lib/include/THC -I/usr/local/cuda/include -I/local/ady/anaconda3/include/python3.6m complex_mul_kernel_stride_no_permute.cu -o complex_mul_kernel_stride_no_permute.out -std=c++11
Segmentation fault
*/
//int main(void)
//{
//    int N = 1;
//    int F = 1;
//    int C = 4;
//    int H = 16;
//    int W = 8;
//    int size_input = N * C * H * W * 2;
//    int size_filter = F * C * H * W * 2;
//    int size_output = N * F * H * W * 2;
//    int cuda_block_threads = 32;
//
//    // auto dims = {128, 32, 16, 8, 2};
//    //    at::Tensor x = at::randn({128, 32, 16, 8, 2});
//    //    at::Tensor y = at::randn({128, 32, 16, 8, 2});
//    //    at::Tensor out = at::zeros({128, 32, 16, 8, 2});
//    float *x, *y, * out;
//
//    // Allocate unified memory - accessible from cpu or gpu
//    hipMallocManaged(&x, size_input*sizeof(float));
//    hipMallocManaged(&y, size_filter*sizeof(float));
//    hipMallocManaged(&out, size_output*sizeof(float));
//
//    for (int i=0; i < size_input-1; i+=2) {
//        x[i] = -8;
//        x[i+1] = -1;
//        y[i] = -1;
//        y[i+1] = -2;
//        out[i] = 0.0f;
//        out[i+1] = 0.0f;
//    }
//
//    const dim3 blocks(N, F);
//
//    // for(int i=0; i<32; ++i)
//    complex_mul_cuda_kernel<float><<<blocks, cuda_block_threads>>>(
//        x, y, out, N, F, C, H, W);
//
//    hipFree(x);
//    hipFree(y);
//    hipFree(out);
//
//    printf("finished computation\n");
//
//    return 0;
//}