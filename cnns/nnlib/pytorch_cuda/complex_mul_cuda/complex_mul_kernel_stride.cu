#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>

namespace {

template <typename scalar_t>
__device__ __forceinline__ void single_mul(
    scalar_t x_re,
    scalar_t x_im,
    scalar_t y_re,
    scalar_t y_im,
    scalar_t* out_re,
    scalar_t* out_im) {

    scalar_t uavc = x_re * (y_re + y_im);
    *out_re += uavc - (x_re + x_im) * y_im;
    *out_im += (x_im - x_re) * y_re + uavc;
}


template <typename scalar_t>
__global__ void complex_mul_cuda_kernel(
    const scalar_t* __restrict__ x,
    const scalar_t* __restrict__ y,
    scalar_t* __restrict__ out,
    const int N, const int F, const int C, const int H, const int W) {

    const int I = 2; // the last dimension for the complex number
    const int batch_size = H * W * C * I;  // size of the plan from the batch

    const int n = blockIdx.x; // current data point in the batch
    const int f = blockIdx.y; // current filter from the filter bank
    const int start = threadIdx.x; // current column to be computed
    const int raw_stride = blockDim.x;  // stride for the H*W map is equal to the number of threads declared in a block

    const int n_idx = n * batch_size;  // start index in the batch for this input map
    const int f_idx = f * batch_size;  // start index in the bank for this filter

    // find index for the output
    const int no_idx = n * F*H*W*I; // output index for the batch data point
    const int fo_idx = f * H*W*I;   // output index for the filter/channel

    const int plane_size = H*W;

    for (int raw_pixel = start; raw_pixel < plane_size; raw_pixel += raw_stride)  {
        /* If the plane is of size HxW = 32x32, and the raw_pixel is 45, then
        the h is 45 / 32 = 1, and the W (current column) is 45 - 1*32 = 13. */
        const int h = raw_pixel / plane_size;  // current row in the H,W,C,I plane
        const int w = raw_pixel - h * H;       // current col in the W,C,I linear space
        const int h_idx = h * W * C * I;   // start index for this row
        const int w_idx = w * C * I;       // start index for this column

        // index in the input map
        const int N_idx = n_idx + h_idx + w_idx; // index for this C,I component in input

        // index in the filter
        const int F_idx = f_idx + h_idx + w_idx; // index for this C,I component in filter

        // find the final index (last mile) for the output
        const int ho_idx = h * W * I;   // output index for row
        const int wo_idx = w * I;       // output index for col
        const int O_idx = no_idx + fo_idx + ho_idx + wo_idx;

        scalar_t out_re = 0.0;
        scalar_t out_im = 0.0;

        for (int c = 0; c < C; ++c) {
            scalar_t x_re = x[N_idx + c*I];
            scalar_t x_im = x[N_idx + c*I + 1];
            scalar_t y_re = y[F_idx + c*I];
            scalar_t y_im = y[F_idx + c*I + 1];
            single_mul(x_re, x_im, y_re, y_im, &out_re, &out_im);
        }
        out[O_idx] = out_re;
        out[O_idx + 1] = out_im;
    }
}

} // namespace

void complex_mul_stride_cuda(at::Tensor x, at::Tensor y, at::Tensor out) {

    const int threads = 1024;  // corresponds to W

    const auto N = x.size(0);  // batch_size
    const auto F = y.size(0);  // filter_bank_size
    const auto C = x.size(1);  // number of channels
    const auto H = x.size(2);  // height of the matrix
    const auto W = x.size(3);  // width of the matrix

    // set channel as the last but one dimension
    x = x.permute({0,2,3,1,4});
    y = y.permute({0,2,3,1,4});

    const auto x_blocks = N;
    const auto y_blocks = F;
    const dim3 blocks(x_blocks, y_blocks);

    AT_DISPATCH_FLOATING_TYPES(x.type(), "complex_mul_cuda",
    ([&] {
        complex_mul_cuda_kernel<scalar_t><<<blocks, threads>>>(
        x.data<scalar_t>(), y.data<scalar_t>(), out.data<scalar_t>(),
        N, F, C, H, W);
    }));

    // restore the channel to the second dimension
    x = x.permute({0,3,1,2,4});
    y = y.permute({0,3,1,2,4});
}
