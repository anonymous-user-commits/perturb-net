//  Band-limited CNNs
//  Copyright (c) 2019. Adam Dziedzic
//  Licensed under The Apache License [see LICENSE for details]
//  Written by Adam Dziedzic

#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <string>

namespace {

void complex_mul_cublas(
    at::Tensor x,
    at::Tensor y,
    at::Tensor out) {

    const auto H = x.size(0);
    const auto W = x.size(1);
    const auto N = x.size(2);  // batch_size
    const auto C = x.size(3);  // number of channels
    const auto F = y.size(2);  // filter_bank_size

    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        throw "CUBLAS initialization failed\n";
    }

    hipblasOperation_t transa = HIPBLAS_OP_N; // 	the non-transpose operation is selected
    hipblasOperation_t transb = HIPBLAS_OP_N;

    hipComplex alpha = make_hipComplex(1.0, 0.0);

    hipblasStatus_t stat;
    stat = hipblasStatus_t hipblasCgemm(handle,
                           transa, transb,
                           int m, int n, int k,
                           &alpha,
                           const hipComplex       *A, int lda,
                           const hipComplex       *B, int ldb,
                           const hipComplex       *beta,
                           hipComplex       *C, int ldc);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        hipblasDestroy(handle);
        throw "CUBLAS cGemm failed\n";
    }
}

}

